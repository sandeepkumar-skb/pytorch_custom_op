#include "hip/hip_runtime.h"
#include <torch/script.h>
#include "pyt_all_reduce_kernel.hh"
#define BLOCKX_DIM 256

void cpu_all_reduce(int* sum, int* data, int n){
    int temp_sum = 0;
    for (int i=0; i<n; ++i){
        temp_sum += data[i];
    }
    *sum = temp_sum;
}

__global__
void gpu_all_reduce(int *sum, int* data, int n){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    int temp = 0;
    for (int i=idx; i < n; i += stride){
        temp += data[i];
    }

    atomicAdd(sum, temp);
}


torch::Tensor all_reduce_launcher(torch::Tensor input){
    torch::Device device(torch::kCUDA, 0);
    torch::Tensor output = torch::zeros(1, torch::kInt);
    if (input.device() == device){
        output = output.to(device);
        dim3 blockSize(BLOCKX_DIM);
        dim3 gridSize((input.size(0)+BLOCKX_DIM-1)/BLOCKX_DIM);
        const hipStream_t stream = at::cuda::getCurrentCUDAStream();
        gpu_all_reduce<<<gridSize, blockSize, 0, stream>>>(output.data_ptr<int>(), 
                input.data_ptr<int>(), 
                input.size(0));
    }
    else{
        cpu_all_reduce(output.data_ptr<int>(), input.data_ptr<int>(), input.size(0));
    }
    return output;
           
}

