#include "hip/hip_runtime.h"
#include <torch/script.h>
#include "pyt_all_reduce_kernel.hh"
#define BLOCKX_DIM 256

template<typename scalar_t>
void cpu_all_reduce(float * sum, scalar_t* data, int n){
    scalar_t temp_sum = 0;
    for (int i=0; i<n; ++i){
        temp_sum += data[i];
    }
    *sum = temp_sum;
}

template<typename scalar_t>
__global__
void gpu_all_reduce(float *sum, scalar_t *data, int n){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    scalar_t temp = 0;
    for (int i=idx; i < n; i += stride){
        temp += data[i];
    }

    atomicAdd(sum, temp);
}


torch::Tensor all_reduce_launcher(torch::Tensor input){
    torch::Device device(torch::kCUDA, 0);
    torch::Tensor output = torch::zeros(1, torch::kFloat);
    if (input.device() == device){
        output = output.to(device);
        dim3 blockSize(BLOCKX_DIM);
        dim3 gridSize((input.size(0)+BLOCKX_DIM-1)/BLOCKX_DIM);
        const hipStream_t stream = at::cuda::getCurrentCUDAStream();
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "gpu_all_reduce", ([&] {
            gpu_all_reduce<scalar_t><<<gridSize, blockSize, 0, stream>>>(output.data_ptr<float>(), 
                input.data_ptr<scalar_t>(), 
                input.size(0));
        } ));
    }
    else{
        cpu_all_reduce<int>(output.data_ptr<float>(), input.data_ptr<int>(), input.size(0));
    }
    return output;
}

